#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

/**************************************************************************
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o PW_Cracking_Cuda PW_Cracking_Cuda.cu
    ./PW_Cracking_Cuda
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "CV78";
  char plain_password2[] = "FR83";
  char plain_password3[] = "IS53";
  char plain_password4[] = "TB35";
			

  char *a = attempt;
  char *b = attempt;
  char *c = attempt;
  char *d = attempt;
  
  char *p1 = plain_password1;
  char *p2 = plain_password2;
  char *p3 = plain_password3;
  char *p4 = plain_password4;
  
  
  while(*a == *p1) {
    if(*a == '\0') {
       printf("Password Found: %s\n",plain_password1);
    break;
    }
    a++;
    p1++;
  }
  while(*b == *p2) {
    if(*b == '\0') {
      printf("Password Found: %s\n",plain_password2);
    break;
    }
    b++;
    p2++;
  }
  while(*c == *p3) {
    if(*c == '\0') {
     printf("Password Found: %s\n",plain_password3);
    break;
    }
    c++;
    p3++;
  }
  while(*d == *p4) {
    if(*d == '\0') {
      printf("Password Found: %s\n",plain_password4);
    break;
    }
    d++;
    p4++;
  }
  return 0;
}

/****************************************************************************
  The kernel function assume that there will be only one thread and uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
  char a, b;
  int w, y, x, z;
  
  char password[8];
  password[6] = '\0'; 
  a = blockIdx.x+65;
  b = threadIdx.x+65;
  
  password[0] =a;
  password[1] =b;
  
  
  for(w=48; w<=57; w++){
    for(x=48; x<=57; x++){
     for(y=48; y<=57; y++){
      for(z=48; z<=57; z++){
	password[3] = w;
        password[2] = x;
        password[4] = y;
        password[5] = z;

        if(is_a_match(password)) {
        printf("password found: %s\n", password);
      } else {
        //printf("tried: %s\n", password);		  
      }
}
}
}
}

}
int timediff(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main(){

  struct timespec start, finish;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26, 26>>>();
  hipDeviceSynchronize();

  
  
  
  

  clock_gettime(CLOCK_MONOTONIC, &finish);
  timediff(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, 
	(time_elapsed/1.0e9));

  return 0;
}


